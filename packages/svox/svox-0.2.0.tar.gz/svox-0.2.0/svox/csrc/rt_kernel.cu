#include "hip/hip_runtime.h"
/*
 * Copyright Alex Yu 2021
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice,
 * this list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include <cstdint>
#include "common.cuh"

#define CUDA_N_THREADS 256

namespace {
namespace device {
// SH Coefficients from https://github.com/google/spherical-harmonics
__device__ __constant__ const float C0 = 0.28209479177387814;
__device__ __constant__ const float C1 = 0.4886025119029199;
__device__ __constant__ const float C2[] = {
    1.0925484305920792,
    -1.0925484305920792,
    0.31539156525252005,
    -1.0925484305920792,
    0.5462742152960396
};

__device__ __constant__ const float C3[] = {
    -0.5900435899266435,
    2.890611442640554,
    -0.4570457994644658,
    0.3731763325901154,
    -0.4570457994644658,
    1.445305721320277,
    -0.5900435899266435
};

template <typename scalar_t>
__device__ __inline__ void _precalc_sh(
    const int order,
    const scalar_t* __restrict__ dir,
    scalar_t* __restrict__ out_mult) {

    out_mult[0] = C0;
    const scalar_t x = dir[0], y = dir[1], z = dir[2];
    out_mult[1] = - C1 * y;
    out_mult[2] = C1 * z;
    out_mult[3] = -C1 * x;
    if (order > 1) {
        const scalar_t xx = x * x, yy = y * y, zz = z * z;
        out_mult[4] = C2[0] * x * y;
        out_mult[5] = C2[1] * y * z;
        out_mult[6] = C2[2] * (2.0 * zz - xx - yy);
        out_mult[7] = C2[3] * x * z;
        out_mult[8] = C2[4] * (xx - yy);
        if (order > 2) {
            const scalar_t tmp_zzxxyy = 4 * zz - xx - yy;
            out_mult[9] = C3[0] * y * (3 * xx - yy);
            out_mult[10] = C3[1] * x * y * z;
            out_mult[11] = C3[2] * y * tmp_zzxxyy;
            out_mult[12] = C3[3] * z * (2 * zz - 3 * xx - 3 * yy);
            out_mult[13] = C3[4] * x * tmp_zzxxyy;
            out_mult[14] = C3[5] * z * (xx - yy);
            out_mult[15] = C3[6] * x * (xx - 3 * yy);
        }
    }
}

template <typename scalar_t>
__device__ __inline__ void _dda_unit(
        const scalar_t* __restrict__ cen,
        const scalar_t* __restrict__ _invdir,
        scalar_t* __restrict__ tmin,
        scalar_t* __restrict__ tmax) {
    scalar_t t1, t2;
    *tmin = 0.0f;
    *tmax = 1e9f;
#pragma unroll
    for (int i = 0; i < 3; ++i) {
        t1 = - cen[i] * _invdir[i];
        t2 = t1 +  _invdir[i];
        *tmin = max(*tmin, min(t1, t2));
        *tmax = min(*tmax, max(t1, t2));
    }
}


template <typename scalar_t>
__device__ __inline__ void trace_ray(
    const torch::PackedTensorAccessor32<scalar_t, 5, torch::RestrictPtrTraits>
        data,
    const torch::PackedTensorAccessor32<int32_t, 4, torch::RestrictPtrTraits>
        child,
        const scalar_t* __restrict__ origin,
        const scalar_t* __restrict__ dir,
        const scalar_t* __restrict__ vdir,
        scalar_t step_size,
        scalar_t stop_thresh,
        scalar_t background_brightness,
        int sh_order,
        float sigma_scale,
        torch::TensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, int32_t> out) {

    scalar_t tmin, tmax;
    scalar_t invdir[3];
    const int tree_N = child.size(1);
    const int data_dim = data.size(4);
    const int out_data_dim = out.size(0);

#pragma unroll
    for (int i = 0; i < 3; ++i) {
        invdir[i] = 1.0 / (dir[i] + 1e-9);
    }
    _dda_unit(origin, invdir, &tmin, &tmax);

    const int rgb_dim = out_data_dim - 1;
    if (tmax < 0 || tmin > tmax) {
        // Ray doesn't hit box
        for (int j = 0; j < rgb_dim; ++j) {
            out[j] = background_brightness;
        }
        out[rgb_dim] = 0.f;  // Alpha
        return;
    } else {
        for (int j = 0; j < out_data_dim; ++j) {
            out[j] = 0.f;
        }
        scalar_t pos[3], tmp;
        scalar_t sh_mult[16];
        if (sh_order >= 0) {
            _precalc_sh<scalar_t>(sh_order, vdir, sh_mult);
        }

        scalar_t light_intensity = 1.f;
        scalar_t t = tmin;
        const int n_coe = (sh_order + 1) * (sh_order + 1);
        scalar_t cube_sz;
        while (t < tmax) {
            for (int j = 0; j < 3; ++j) {
                pos[j] = origin[j] + t * dir[j];
            }

            scalar_t* tree_val = query_single_from_root<scalar_t>(data, child,
                        pos, &cube_sz);

            scalar_t att;
            scalar_t subcube_tmin, subcube_tmax;
            _dda_unit(pos, invdir, &subcube_tmin, &subcube_tmax);

            const scalar_t t_subcube = (subcube_tmax - subcube_tmin) / cube_sz;
            const scalar_t delta_t = t_subcube + step_size;
            const scalar_t sigma = tree_val[data_dim - 1];
            if (sigma > 0.f) {
                att = expf(-delta_t * sigma * sigma_scale);
                const scalar_t weight = light_intensity * (1.f - att);

                if (sh_order >= 0) {
                    for (int t = 0; t < rgb_dim; ++ t) {
                        int off = t * n_coe;
                        tmp = sh_mult[0] * tree_val[off] +
                            sh_mult[1] * tree_val[off + 1] +
                            sh_mult[2] * tree_val[off + 2];
                        for (int i = 3; i < n_coe; ++i) {
                            tmp += sh_mult[i] * tree_val[off + i];
                        }
                        out[t] += weight / (1.f + expf(-tmp));
                    }
                } else {
                    for (int j = 0; j < rgb_dim; ++j) {
                        out[j] += tree_val[j] * weight;
                    }
                }
                out[rgb_dim] += weight;
                light_intensity *= att;

                if (light_intensity < stop_thresh) {
                    // Almost full opacity, stop
                    scalar_t scale = 1.0 / (1.0 - light_intensity);
                    for (int j = 0; j < rgb_dim; ++j) {
                        out[j] *= scale;
                    }
                    out[rgb_dim] = 1.f;  // Alpha
                    return;
                }
            }
            t += delta_t;
        }
        for (int j = 0; j < rgb_dim; ++j) {
            out[j] += light_intensity * background_brightness;
        }
    }
}

template <typename scalar_t>
__global__ void render_ray_kernel(
    const torch::PackedTensorAccessor32<scalar_t, 5, torch::RestrictPtrTraits>
        data,
    const torch::PackedTensorAccessor32<int32_t, 4, torch::RestrictPtrTraits>
        child,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits>
        origins,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits>
        dirs,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits>
        vdirs,
    scalar_t step_size,
    scalar_t stop_thresh,
    scalar_t background_brightness,
    int sh_order,
    const scalar_t* __restrict__ offset,
    const scalar_t* __restrict__ invradius,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits>
        out
        ) {
    CUDA_GET_THREAD_ID(tid, origins.size(0));
    scalar_t origin[3] = {origins[tid][0], origins[tid][1], origins[tid][2]};
    transform_coord<scalar_t>(origin, offset, invradius);
    scalar_t sigma_scale = 1.0 / *invradius;
    trace_ray<scalar_t>(
        data, child,
        origin,
        &dirs[tid][0],
        &vdirs[tid][0],
        step_size,
        stop_thresh,
        background_brightness,
        sh_order,
        sigma_scale,
        out[tid]);
}


template <typename scalar_t>
__global__ void render_image_kernel(
    const torch::PackedTensorAccessor32<scalar_t, 5, torch::RestrictPtrTraits>
        data,
    const torch::PackedTensorAccessor32<int32_t, 4, torch::RestrictPtrTraits>
        child,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits>
        c2w,
    scalar_t step_size,
    scalar_t stop_thresh,
    scalar_t background_brightness,
    int sh_order,
    float fx,
    float fy,
    int width,
    int height,
    const scalar_t* __restrict__ offset,
    const scalar_t* __restrict__ invradius,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits>
        out
        ) {
    CUDA_GET_THREAD_ID(tid, width * height);
    int iy = tid / width, ix = tid % width;
    scalar_t x = (ix - 0.5 * width) / fx;
    scalar_t y = (iy - 0.5 * height) / fy;
    scalar_t z = sqrtf(x * x + y * y + 1.0);
    x /= z; y /= z; z = 1.0f / z;

    scalar_t dir[3];
    dir[0] = c2w[0][0] * x + c2w[0][1] * y + c2w[0][2] * z;
    dir[1] = c2w[1][0] * x + c2w[1][1] * y + c2w[1][2] * z;
    dir[2] = c2w[2][0] * x + c2w[2][1] * y + c2w[2][2] * z;
    scalar_t origin[3] = {c2w[0][3], c2w[1][3], c2w[2][3]};

    transform_coord<scalar_t>(origin, offset, invradius);
    scalar_t sigma_scale = 1.0 / *invradius;
    trace_ray<scalar_t>(
        data, child,
        origin,
        dir,
        dir,
        step_size,
        stop_thresh,
        background_brightness,
        sh_order,
        sigma_scale,
        out[iy][ix]);
}

}  // namespace device


// Compute RGBA output dimension from input dimension & SH order
__host__ int get_out_data_dim(int sh_order, int in_data_dim) {
    int out_data_dim;
    if (sh_order >= 0) {
        const int n_coe = (sh_order + 1) * (sh_order + 1);
        out_data_dim = (in_data_dim - 1) / n_coe + 1;
    } else {
        out_data_dim = in_data_dim;
    }
    return out_data_dim;
}

}  // namespace

torch::Tensor _volume_render_cuda(torch::Tensor data, torch::Tensor child,
                            torch::Tensor origins, torch::Tensor dirs,
                            torch::Tensor vdirs, torch::Tensor offset,
                            torch::Tensor invradius, float step_size,
                            float stop_thresh, float background_brightness,
                            int sh_order) {
    const auto Q = origins.size(0);

    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, CUDA_N_THREADS);
    int out_data_dim = get_out_data_dim(sh_order, data.size(4));
    torch::Tensor result = torch::zeros({Q, out_data_dim}, origins.options());
    AT_DISPATCH_FLOATING_TYPES(origins.type(), __FUNCTION__, [&] {
            device::render_ray_kernel<scalar_t><<<blocks, CUDA_N_THREADS>>>(
                data.packed_accessor32<scalar_t, 5, torch::RestrictPtrTraits>(),
                child.packed_accessor32<int32_t, 4, torch::RestrictPtrTraits>(),
                origins.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                dirs.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                vdirs.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                step_size,
                stop_thresh,
                background_brightness,
                sh_order,
                offset.data<scalar_t>(),
                invradius.data<scalar_t>(),
                result.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>());
    });
    CUDA_CHECK_ERRORS;
    return result;
}

torch::Tensor _volume_render_image_cuda(
    torch::Tensor data, torch::Tensor child, torch::Tensor offset,
    torch::Tensor invradius, torch::Tensor c2w, float fx, float fy, int width,
    int height, float step_size, float stop_thresh,
    float background_brightness, int sh_order) {
    const size_t Q = size_t(width) * height;

    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, CUDA_N_THREADS);
    int out_data_dim = get_out_data_dim(sh_order, data.size(4));
    torch::Tensor result = torch::zeros({height, width, out_data_dim}, data.options());

    AT_DISPATCH_FLOATING_TYPES(data.type(), __FUNCTION__, [&] {
            device::render_image_kernel<scalar_t><<<blocks, CUDA_N_THREADS>>>(
                data.packed_accessor32<scalar_t, 5, torch::RestrictPtrTraits>(),
                child.packed_accessor32<int32_t, 4, torch::RestrictPtrTraits>(),
                c2w.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                step_size,
                stop_thresh,
                background_brightness,
                sh_order,
                fx,
                fy,
                width,
                height,
                offset.data<scalar_t>(),
                invradius.data<scalar_t>(),
                result.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>());
    });
    CUDA_CHECK_ERRORS;
    return result;
}
