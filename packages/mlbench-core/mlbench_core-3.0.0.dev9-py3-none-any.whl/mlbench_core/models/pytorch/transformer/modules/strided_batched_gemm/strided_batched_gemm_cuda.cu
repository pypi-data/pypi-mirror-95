#include <vector>
#include <iostream>

#include <ATen/ATen.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>

#include "THC/THC.h"
#include "THC/THCBlas.h"

// symbol to be automatically resolved by PyTorch libs
extern THCState *state;


at::Tensor strided_batched_gemm_cuda(
    float beta,
    at::Tensor in_result,
    float alpha,
    at::Tensor batch1,
    at::Tensor batch2) {

  bool transpose_result;
  char transpose_batch1, transpose_batch2;
  int64_t lda, ldb, ldc;
  at::Tensor result, input1, input2;
  if (in_result.stride(1) == 1)
  {
    transpose_result = false;
    result = in_result;
    ldc = result.stride(2);
  }
  else if (in_result.stride(2) == 1)
  {
    transpose_result = true;

    at::Tensor swap = batch2;
    batch2 = batch1;
    batch1 = swap;

    result = in_result;
    ldc = result.stride(1);
  } else {
    AT_ASSERTM(false, "result should be contiguous");
  }

  if (batch1.stride(transpose_result ? 2 : 1) == 1 &&
      batch1.stride(transpose_result ? 1 : 2) != 0) {
    transpose_batch1 = 'n';
    input1 = batch1;
    lda = input1.stride(transpose_result ? 1 : 2);
  } else if (batch1.stride(transpose_result ? 1 : 2) == 1 &&
             batch1.stride(transpose_result ? 2 : 1) != 0) {
    transpose_batch1 = 't';
    input1 = batch1;
    lda = input1.stride(transpose_result ? 2 : 1);
  } else {
    AT_ASSERTM(false, "input1 should be contiguous");
  }

  if (batch2.stride(transpose_result ? 2 : 1) == 1 &&
      batch2.stride(transpose_result ? 1 : 2) != 0) {
    transpose_batch2 = 'n';
    input2 = batch2;
    ldb = input2.stride(transpose_result ? 1 : 2);
  } else if (batch2.stride(transpose_result ? 1 : 2) == 1 &&
             batch2.stride(transpose_result ? 2 : 1) != 0) {
    transpose_batch2 = 't';
    input2 = batch2;
    ldb = input2.stride(transpose_result ? 2 : 1);
  } else {
    AT_ASSERTM(false, "input2 should be contiguous");
  }
  int64_t num_batches = result.size(0);

  THCudaBlas_HgemmStridedBatched(
      state,
      transpose_batch1,
      transpose_batch2,
      result.size(transpose_result ? 2 : 1),
      result.size(transpose_result ? 1 : 2),
      input1.size(transpose_result ? 1 : 2),
      alpha,
      static_cast<const c10::Half *>(input1.data_ptr()), lda, input1.stride(0),
      static_cast<const c10::Half *>(input2.data_ptr()), ldb, input2.stride(0),
      beta,
      static_cast<c10::Half *>(result.data_ptr()), ldc, result.stride(0),
      num_batches);

  return in_result;
}


